// CUDA przykład (c) Andrzej Łukaszewski 2010
// Dodawanie macierzy na GPU:                   kompilacja: nvcc addmat.cu 

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void AddMatrixKernel1(float *A, float *B, float *C, int N)
{
    int adres = threadIdx.x + N * blockIdx.x;
    C[adres] = A[adres] + B[adres];
}

void GPUMatrixAdd(float *A, float *B, float *C, int N)
{
    int size = N*N*sizeof(float);
    float  *Ad, *Bd, *Cd;   //  macierze na GPU

    hipMalloc(&Ad, size);
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMalloc(&Bd, size);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    hipMalloc(&Cd, size);

    // Wywołanie jądra   np.:
    AddMatrixKernel1<<<N,N>>>(Ad,Bd,Cd,N); // N bloków po N wątków

    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    hipFree(Ad);   hipFree(Bd);   hipFree(Cd);
}

int main()
{
    float  A[4]={ 1., 1., 1., 1. };
    float  B[4]={ 2., 3., 4., 5. };
    float  C[4];

    printf("A[1]=%f\n", A[1]);  printf("B[1]=%f\n", B[1]);
    GPUMatrixAdd(A,B,C,2);
    printf("C[1]=%f\n", C[1]);
    return 0;
}
